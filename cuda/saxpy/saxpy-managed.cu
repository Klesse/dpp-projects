

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(int n, int a, float *x, float*y){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n)
    y[index] = x[index] * a  + y[index]; 

}


int main(void){
  int N=10000;
  float *x, *y;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

    for (int i=0;i<N;i++){
    x[i] = 2.0f;
    y[i] = 3.0f;
  }
  printf("First element of y %f\n",y[0]);

  int blockSize = 1024;
  int numberBlocks = (N+blockSize-1)/blockSize;

  saxpy<<<numberBlocks,blockSize>>>(N,10,x,y);

  hipDeviceSynchronize();

  printf("First element of y %f\n",y[0]);

  hipFree(x);
  hipFree(y);

}
