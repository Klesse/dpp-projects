

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(int n, int a, float *d_x, float *d_y){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    d_y[i] = d_x[i] * a + d_y[i];
}


int main(void){
  int N = 10000;
  float *x, *y, *d_x, *d_y;

  x = (float *)malloc(N*sizeof(float));
  y = (float *)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));

  for(int i=0;i<N;i++){
    x[i] = 2;
    y[i] = 3;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 1024;
  int numBlocks = (N+blockSize-1)/blockSize;

  saxpy<<<numBlocks,blockSize>>>(N,10,d_x,d_y);


  hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  printf("First element of y: %f",y[0]);

  hipFree(d_x);
  hipFree(d_y);

}
