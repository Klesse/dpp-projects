
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void mul(float *d_A, float *d_B, float *d_C, int n_elem){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < n_elem && j < n_elem){
      d_C[i*n_elem+j]=0;
      for(int k=0; k<n_elem; k++)
        d_C[i*n_elem+j] += d_A[i*n_elem+k] * d_B[k*n_elem+j];
  }
}


int main(void){
  float *A, *B, *C;
  float *d_A, *d_B, *d_C;

  int n_elem = 10000;

  size_t tam_mat = n_elem * n_elem * sizeof(float);

  A = (float *)malloc(tam_mat);
  B = (float *)malloc(tam_mat);
  C = (float *)malloc(tam_mat);

  for (int i=0; i< n_elem*n_elem; i++){
    A[i] = 0.1;
    B[i] = 0.2;
  }

  hipMalloc((void **)&d_A, tam_mat);
  hipMalloc((void **)&d_B, tam_mat);
  hipMalloc((void **)&d_C, tam_mat);

  hipMemcpy(d_A, A, tam_mat, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, tam_mat, hipMemcpyHostToDevice);

  dim3 block;

  block.x = 16;
  block.y = 16;
  block.z = 1;

  dim3 grid;

  grid.x = (n_elem + block.x -1) / block.x;
  grid.y = (n_elem + block.y -1) / block.y;
  grid.z = 1;

  mul<<<grid, block>>>(d_A, d_B, d_C, n_elem);

  hipDeviceSynchronize();

  hipMemcpy(C, d_C, tam_mat, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  free(A);
  free(B);
  free(C);

  return 0;
  }