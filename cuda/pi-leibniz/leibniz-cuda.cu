#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// Função de kernel CUDA para o cálculo da série de Leibniz
__global__
void iter(int n, int* digits, const long int d){
    // Cálculo do índice da thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Verifica se o índice da thread está dentro do intervalo
    if (i < n){
        int div, mod, remainder, signal, digit;

        // Determina o sinal para os termos alternados na série
        if (i % 2 == 0){
            signal = 1;
        } else {
            signal = -1;
        }

        remainder = 4;

        // Realiza o cálculo da série de Leibniz para cada thread
        for(digit = 0; digit < d + 11 && remainder; ++digit) {
            div = remainder / (2 * i + 1);
            mod = remainder % (2 * i + 1);

            // Usa atomicAdd para atualizar o array compartilhado de forma paralela
            atomicAdd(&digits[digit], signal * div);
            remainder = mod * 10;
        }
    }
}

// Função para calcular pi usando a série de Leibniz com CUDA
void pi(char* output, const long int n, const long int d, int num_threads, int num_blocks) {
    // Aloca memória para o array que armazenará os dígitos
    int *digits = (int *)malloc((d+11)*sizeof(int));
    int *d_digits;
    long int digit, i;

    // Inicializa o array de dígitos com zeros
    for(digit = 0; digit < d + 11; ++digit) {
        digits[digit] = 0;
    }

    // Aloca memória na GPU e copia os dados para a GPU
    hipMalloc(&d_digits, (d+11)*sizeof(int));
    hipMemcpy(d_digits, digits, (d+11)*sizeof(int), hipMemcpyHostToDevice);

    // Inicia o kernel CUDA para o cálculo em paralelo
    iter<<<num_blocks, num_threads>>>(n, d_digits, d);

    // Copia os resultados de volta da GPU para a CPU
    hipMemcpy(digits, d_digits, (d+11)*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_digits);

    // Pós-processamento dos resultados para obter o valor final de pi
    for(i = d + 11 - 1; i > 0; --i) {
        digits[i - 1] += digits[i] / 10;
        digits[i] %= 10;
        if(digits[i] < 0) {
            digits[i - 1]--;
            digits[i] += 10;
        }
    }

    // Ajuste de arredondamento
    if(digits[d + 1] >= 5) {
        ++digits[d];
    }

    // Realiza arredondamento adicional e formatação da saída
    for(i = d; i > 0; --i) {
        digits[i - 1] += digits[i] / 10;
        digits[i] %= 10;
    }

    // Formata a string de saída
    output[0] = digits[0] + '0';
    output[1] = '.';
    for(i = 1; i <= d; i++)
        output[i + 1] = digits[i] + '0';
    output[d + 2] = 0;

    // Libera a memória alocada
    free(digits);
}

// Função principal
int main(int argc, char* argv[]) {
    int n, d, num_threads, num_blocks;

    FILE *in, *out;

    // Abre os arquivos de entrada e saída
    in = fopen("pi.in", "r");
    out = fopen("pi.out", "w");

    // Lê os valores de entrada do arquivo
    fscanf(in, "%d%d%d%d", &n, &d, &num_threads, &num_blocks);

    char output[d + 4];

    // Chama a função pi para calcular e formatar a aproximação de pi
    pi(output, n, d, num_threads, num_blocks);

    // Escreve o resultado no arquivo de saída
    fprintf(out, "%s\n", output);
    fflush(out);

    // Fecha os arquivos
    fclose(in);
    fclose(out);

    return EXIT_SUCCESS;
}
